#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_streebog.h"
}

#include "miner.h"
#include "cuda_helper.h"

#include <stdio.h>
#include <memory.h>

#define NBN 2

// GOST CPU Hash
extern "C" void gostd_hash(void *output, const void *input)
{
	unsigned char _ALIGN(64) hash[64];

	sph_gost512(hash, (const void*)input, 80);
	sph_gost256(hash, (const void*)hash, 64);

	memcpy(output, hash, 32);
}

extern "C" void gostd(void *output, const void *input, size_t len)
{
	unsigned char _ALIGN(64) hash[64];

	sph_gost512(hash, (const void*)input, len);
	sph_gost256(hash, (const void*)hash, 64);

	memcpy(output, hash, 32);
}

//#define _DEBUG
#define _DEBUG_PREFIX "gost"
#include "cuda_debug.cuh"

static bool init[MAX_GPUS] = { 0 };
extern void gostd_init(int thr_id);
extern void gostd_free(int thr_id);
extern void gostd_setBlock_80(uint32_t *pdata, uint32_t *ptarget);
extern void gostd_hash_80(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces);

extern "C" int scanhash_gostd(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	//ptarget[7] = 0x000000FF;
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 25);
	if (init[thr_id]) throughput = min(throughput, (max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x03;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		gostd_init(thr_id);

		init[thr_id] = true;
	}

	for (int k=0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);

	gostd_setBlock_80(endiandata, ptarget);

	do {
		// Hash with CUDA
		*hashes_done = pdata[19] - first_nonce + throughput;

		gostd_hash_80(thr_id, throughput, pdata[19], work->nonces);
		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];

			endiandata[19] = swab32 (work->nonces[0]);
			gostd_hash(vhash, endiandata);
			if (swab32(vhash[0]) <= ptarget[7] /*&& fulltest(vhash, ptarget)*/) 
			{
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != UINT32_MAX) 
				{
					endiandata[19] = swab32 (work->nonces[1]);
					gostd_hash(vhash, endiandata);
					if (swab32(vhash[0]) <= ptarget[7] /*&& fulltest(vhash, ptarget)*/)
					{
						work->valid_nonces++;
						bn_set_target_ratio(work, vhash, 1);
					}
					pdata[19] = max(work->nonces[0], work->nonces[1]);
				} 
				else 
					pdata[19] = work->nonces[0];
				return work->valid_nonces;
			}
			else if (swab32(vhash[0]) > ptarget[7]) 
			{
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;

	return 0;
}

// cleanup
extern "C" void free_gostd(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	gostd_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
